
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <cmath>
#include <array>
using namespace std;

__global__
void test(char *a, char *b, int N) {
    int i = blockIdx.x;
    if (i<N) {
        if(a[i] == 'A')
        {
            b[i] = 'T';
        }
        if(a[i] == 'C')
        {
            b[i] = 'G';
        }
        if(a[i] == 'G')
        {
            b[i] = 'C';
        }
        if(a[i] == 'T')
        {
            b[i] = 'A';
        }
    }
}

int main () {
    ifstream inFS;
    inFS.open("dna.txt");
    string dna;
    getline(inFS, dna);
    int N = dna.length();
    char ha[N], hb[N];
    char *da, *db;
    hipMalloc((void **)&da, N*sizeof(char));
    hipMalloc((void **)&db, N*sizeof(char));
    for (int i = 0; i<N; ++i) {
        ha[i] = dna[i];
    }
    hipMemcpy(da, ha, N*sizeof(char), hipMemcpyHostToDevice);
    test<<<N, 1>>>(da, db, N);
    hipMemcpy(hb, db, N*sizeof(char), hipMemcpyDeviceToHost);
    ofstream outFS;
    outFS.open("output.txt");
    for(int i = 0; i<N; ++i)
    {
      cout << hb[i];
    }
    outFS.close();
    hipFree(da);
    hipFree(db);
    return 0;
}
