#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <cmath>
#include <array>
using namespace std;

int main () {
    ifstream inFS;
    inFS.open("dna.txt");
    string dna;
    getline(inFS, dna);
    int N = dna.length();
    char ha[N], hb[N];
    char *da, *db;
    hipMalloc((void **)&da, N*sizeof(char));
    hipMalloc((void **)&db, N*sizeof(char));
    for (int i = 0; i<N; ++i) {
        ha[i] = dna[i];
    }
    for(int i = 0; i < ha.size(); ++i)
    {
        cout << ha[i];
    }
    cout << endl;
}