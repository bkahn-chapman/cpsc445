#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <array>
#include <string>
using namespace std;

__global__
void squareroot(double *a, double *b, int N) {
    __shared__ int temp;
    int i = blockIdx.x;
    if (i<N) {
        b[i] = sqrt(a[i]);
        temp++;
    }
}

int main () {
    ifstream inFS;
    inFS.open("input.csv");
    //system("head input.csv");
    vector<double> nums;
    string line;
    while(getline(inFS, line))
    {
        nums.push_back(stod(line));
    }
    inFS.close();
    int N = nums.size();
    double ha[N], hb[N];
    double *da, *db;
    hipMalloc((void **)&da, N*sizeof(double));
    hipMalloc((void **)&db, N*sizeof(double));
    for (int i = 0; i<N; ++i) {
        ha[i] = nums[i];
    }
    hipMemcpy(da, ha, N*sizeof(double), hipMemcpyHostToDevice);
    squareroot<<<N, 1>>>(da, db, N);
    hipMemcpy(hb, db, N*sizeof(double), hipMemcpyDeviceToHost);
    ofstream outFS;
    outFS.open("output.csv");
    for(int i = 0; i<N; ++i)
    {
      outFS << hb[i] << endl;
    }
    cout << temp << endl;
    outFS.close();
    hipFree(da);
    hipFree(db);
    return 0;
}