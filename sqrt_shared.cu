#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <array>
#include <string>
using namespace std;

__global__
void squareroot(double *a, int N) {
    __shared__ double hb[N];
    int i = blockIdx.x;
    if (i<N) {
        hb[i] = sqrt(a[i]);
    }
}

int main () {
    ifstream inFS;
    inFS.open("input.csv");
    //system("head input.csv");
    vector<double> nums;
    string line;
    while(getline(inFS, line))
    {
        nums.push_back(stod(line));
    }
    inFS.close();
    int N = nums.size();
    double ha[N];
    double *da;
    hipMalloc((void **)&da, N*sizeof(double));
    for (int i = 0; i<N; ++i) {
        ha[i] = nums[i];
    }
    hipMemcpy(da, ha, N*sizeof(double), hipMemcpyHostToDevice);
    squareroot<<<N, 1>>>(da, N);
    ofstream outFS;
    outFS.open("output.csv");
    for(int i = 0; i<N; ++i)
    {
      outFS << hb[i] << endl;
    }
    outFS.close();
    hipFree(da);
    hipFree(db);
    return 0;
}