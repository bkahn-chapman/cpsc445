
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <cmath>
#include <array>
using namespace std;

__global__
void count(char *a, int *b, int N) {
    int i = blockIdx.x;
    if (i<N) {
        if(a[i] == 'A')
        {
            b[0]++;
        }
        if(a[i] == 'C')
        {
            b[1]++;
        }
        if(a[i] == 'G')
        {
            b[2]++;
        }
        if(a[i] == 'T')
        {
            b[3]++;
        }
    }
}

int main () {
    ifstream inFS;
    inFS.open("dna.txt");
    string dna;
    getline(inFS, dna);
    int N = dna.length();
    char ha[N];
    int hb[4];
    char *da;
    int *db;
    hipMalloc((void **)&da, N*sizeof(char));
    hipMalloc((void **)&db, 4*sizeof(int));
    for (int i = 0; i<N; ++i) {
        ha[i] = dna[i];
    }
    for (int i = 0; i < 4; ++i) {
        hb[i] = 0;
    }
    hipMemcpy(da, ha, N*sizeof(char), hipMemcpyHostToDevice);
    count<<<N, 1>>>(da, db, N);
    hipMemcpy(hb, db, 4*sizeof(int), hipMemcpyDeviceToHost);
    ofstream outFS;
    outFS.open("output.txt");
    char letters[] = {'A', 'C', 'G', 'T'};
    for(int i = 0; i<4; ++i)
    {
      outFS << letters[i] <<  " " << hb[i];
    }
    outFS.close();
    hipFree(da);
    hipFree(db);
    return 0;
}