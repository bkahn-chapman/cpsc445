
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <cmath>
#include <array>
using namespace std;

__global__
void count(char *a, char *b, int N) {
    int i = blockIdx.x;
    if (i<N) {
        if(a[i] == 'A')
        {
            b[i] = '1';
        }
        if(a[i] == 'C')
        {
            b[i] = '2';
        }
        if(a[i] == 'G')
        {
            b[i] = '3';
        }
        if(a[i] == 'T')
        {
            b[i] = '4';
        }
    }
}

int main () {
    ifstream inFS;
    inFS.open("dna.txt");
    string dna;
    getline(inFS, dna);
    inFS.close();
    int N = dna.length();
    char ha[N], hb[N];
    char *da, *db;
    hipMalloc((void **)&da, N*sizeof(char));
    for(int i = 0; i<N; ++i) {
        ha[i] = dna[i];
    }
    hipMemcpy(da, ha, N*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(db, hb, N*sizeof(char), hipMemcpyHostToDevice);
    count<<<N, 1>>>(da, db, N);
    hipMemcpy(hb, db, N*sizeof(char), hipMemcpyDeviceToHost);
    ofstream outFS;
    outFS.open("output.txt");
    for(int i = 0; i<N; ++i)
    {
      outFS << db[i];
    }
    outFS.close();
    hipFree(da);
    hipFree(db);
    return 0;
}