
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <cmath>
#include <array>
using namespace std;

__global__
void invert(char *a, char *b, int N) {
    int i = blockIdx.x;
    if (i<N) {
        if(a[i] == 'A')
        {
            b[i] = '0';
        }
        if(a[i] == 'C')
        {
            b[i] = '1';
        }
        if(a[i] == 'G')
        {
            b[i] = '2';
        }
        if(a[i] == 'T')
        {
            b[i] = '3';
        }
    }
}

int main () {
    ifstream inFS;
    inFS.open("dna.txt");
    string dna;
    getline(inFS, dna);
    inFS.close();
    int N = dna.length();
    char ha[N], hb[N];
    char *da, *db;
    hipMalloc((void **)&da, N*sizeof(char));
    hipMalloc((void **)&db, N*sizeof(char));
    for (int i = 0; i<N; ++i) {
        ha[i] = dna[i];
    }
    hipMemcpy(da, ha, N*sizeof(char), hipMemcpyHostToDevice);
    invert<<<N, 1>>>(da, db, N);
    hipMemcpy(hb, db, N*sizeof(char), hipMemcpyDeviceToHost);
    ofstream outFS;
    outFS.open("output.txt");
    int aCnt;
    int cCnt;
    int gCnt;
    int tCnt;
    for(int i = 0; i<N; ++i)
    {
        if(hb[i] == '0')
        {
            aCnt++;
        }
        if(hb[i] == '1')
        {
            cCnt++;
        }
        if(hb[i] == '2')
        {
            gCnt++;
        }
        if(hb[i] == '3')
        {
            tCnt++;
        }
    }
    outFS << "A " << aCnt << endl;
    outFS << "T " << tCnt << endl;
    outFS << "G " << gCnt << endl;
    outFS << "C " << cCnt << endl;
    outFS.close();
    hipFree(da);
    hipFree(db);
    return 0;
}
