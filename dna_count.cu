
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <cmath>
#include <array>
using namespace std;

__global__
void count(char *a, char *b, int N) {
    int i = blockIdx.x;
    if (i<N) {
        if(a[i] == 'A')
        {
            b[i] = 'A';
        }
        if(a[i] == 'C')
        {
            b[i] = 'C';
        }
        if(a[i] == 'G')
        {
            b[i] = 'G';
        }
        if(a[i] == 'T')
        {
            b[i] = 'T';
        }
    }
}

int main () {
    ifstream inFS;
    inFS.open("dna.txt");
    string dna;
    getline(inFS, dna);
    inFS.close();
    int N = dna.length();
    char ha[N], hb[N];
    char *da, *db;
    hipMalloc((void **)&da, N*sizeof(char));
    hipMalloc((void **)&db, N*sizeof(char));
    for (int i = 0; i<N; ++i) {
        ha[i] = dna[i];
    }
    hipMemcpy(da, ha, N*sizeof(char), hipMemcpyHostToDevice);
    count<<<N, 1>>>(da, db, N);
    hipMemcpy(hb, db, N*sizeof(char), hipMemcpyDeviceToHost);
    ofstream outFS;
    outFS.open("output.txt");
    int aCnt = 0;
    int cCnt = 0;
    int gCnt = 0;
    int tCnt = 0;
    for(int i = 0; i<N; ++i)
    {
        if(hb[i] == 'A')
        {
            aCnt++;
        }
        if(hb[i] == 'C')
        {
            cCnt++;
        }
        if(hb[i] == 'G')
        {
            gCnt++;
        }
        if(hb[i] == 'T')
        {
            tCnt++;
        }
    }
    outFS << "A " << aCnt << endl;
    outFS << "T " << tCnt << endl;
    outFS << "G " << gCnt << endl;
    outFS << "C " << cCnt << endl;
    outFS.close();
    hipFree(da);
    hipFree(db);
    return 0;
}
