#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <cmath>
#include <array>
using namespace std;

__global__
void count(char *a, int *b, int N) {
    int i = blockIdx.x;
    if (i<N) {
        if(a[i] == 'A')
        {
            b[i] = 0;
        }
        if(a[i] == 'C')
        {
            b[i] = 1;
        }
        if(a[i] == 'G')
        {
            b[i] = 2;
        }
        if(a[i] == 'T')
        {
            b[i] = 3;
        }
    }
}

int main () {
    ifstream inFS;
    inFS.open("dna.txt");
    string dna;
    getline(inFS, dna);
    inFS.close();
    int N = dna.length();
    char ha[N];
    int hb[N];
    char *da;
    int *db;
    hipMalloc((void **)&da, N*sizeof(char));
    hipMalloc((void **)&db, N*sizeof(int));
    for(int i = 0; i<N; ++i) {
        ha[i] = dna[i];
    }
    for(int i = 0; i<N; ++i)
    {
        db[i] = 0;
    }
    cout << endl;
    hipMemcpy(da, ha, N*sizeof(char), hipMemcpyHostToDevice);
    count<<<N, 1>>>(da, db, N);
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);
    ofstream outFS;
    outFS.open("output.txt");
    for(int i = 0; i<hb.size(); ++i)
    {
      outFS << hb[i];
    }
    outFS.close();
    hipFree(da);
    hipFree(db);
    return 0;
}