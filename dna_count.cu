#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <cmath>
#include <array>
using namespace std;

extern __shared__ int results[4];

__global__
void count(char *a, int *b, int N) {
    int i = blockIdx.x;
    int test[4];
    for(int i = 0; i < 4; ++i)
    {
        test[i] = 0;
    }
    if (i<N) {
        if(a[i] == 'A')
        {
            test[0]++;
        }
        if(a[i] == 'C')
        {
            test[1]++;
        }
        if(a[i] == 'G')
        {
            test[2]++;
        }
        if(a[i] == 'T')
        {
            test[3]++;
        }
    }

    __syncthreads();


}

int main () {
    ifstream inFS;
    inFS.open("dna.txt");
    string dna;
    getline(inFS, dna);
    inFS.close();
    int N = dna.length();
    char ha[N];
    int hb[N];
    char *da;
    int *db;
    hipMalloc((void **)&da, N*sizeof(char));
    hipMalloc((void **)&db, N*sizeof(int));
    for(int i = 0; i<N; ++i) {
        ha[i] = dna[i];
        hb[i] = 0;
    }
    hipMemcpy(da, ha, N*sizeof(char), hipMemcpyHostToDevice);
    count<<<N, 1>>>(da, db, N);
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);
    ofstream outFS;
    outFS.open("output.txt");
    for(int i = 0; i<N; ++i)
    {
      outFS << db[i];
    }
    outFS.close();
    hipFree(da);
    hipFree(db);
    return 0;
}