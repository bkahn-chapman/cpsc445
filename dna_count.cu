#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <cmath>
#include <array>
using namespace std;

//test

__global__
void count(char *a, int *b, int N) {
    int i = blockIdx.x;
    __shared__ int share[4];
    if (i<N) {
        if(a[i] == 'A')
        {
            share[0]++;
        }
        if(a[i] == 'C')
        {
            share[1]++;
        }
        if(a[i] == 'G')
        {
            share[2]++;
        }
        if(a[i] == 'T')
        {
            share[3]++;
        }
    }
    __syncthreads();
    for(int t = 0; t < 4; ++t)
    {
        b[t] = shared[t];
    }
}

int main () {
    ifstream inFS;
    inFS.open("dna.txt");
    string dna;
    getline(inFS, dna);
    inFS.close();
    int N = dna.length();
    char ha[N];
    int hb[4];
    char *da;
    int *db;
    hipMalloc((void **)&da, N*sizeof(char));
    hipMalloc((void **)&db, N*sizeof(int));
    for (int i = 0; i<N; ++i) {
        ha[i] = dna[i];
    }
    hipMemcpy(da, ha, N*sizeof(char), hipMemcpyHostToDevice);
    count<<<N, 1>>>(da, db, N);
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);
    ofstream outFS;
    outFS.open("output.txt");
    for(int i = 0; i<4; ++i)
    {
      outFS << share[i];
    }
    outFS.close();
    hipFree(da);
    hipFree(db);
    return 0;
}