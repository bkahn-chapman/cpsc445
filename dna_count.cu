#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <cmath>
#include <array>
using namespace std;

__global__
void count(char *a, int *b, int N) {
    int stride = blockDim.x * 2;
    int tid = threadIdx.x;
    for(int i=tid+W; i<N; i+=W)
    {
        if (i<N) {
            if(a[i] == 'A')
            {
                b[tid]++;
            }
            if(a[i] == 'C')
            {
                b[tid]++;
            }
            if(a[i] == 'G')
            {
                b[tid]++;
            }
            if(a[i] == 'T')
            {
                b[tid]++;
            }
        }
    }
}

int main () {
    ifstream inFS;
    inFS.open("dna.txt");
    string dna;
    getline(inFS, dna);
    inFS.close();
    int N = dna.length();
    char ha[N];
    char *da;
    int *hb = new int[4];
    int *db;
    hipMalloc((void **)&da, N*sizeof(char));
    hipMalloc((void **)&db, N*sizeof(int));
    for (int i = 0; i<N; ++i) {
        ha[i] = dna[i];
    }
    hipMemcpy(da, ha, N*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(db, hb, N*sizeof(int), hipMemcpyHostToDevice);
    int W = 4;
    count<<<1,W>>>(da, db, N);
    hipDeviceSynchronize();
    int sums[4];
    hipMemcpy(sums, da, W*sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0; i < 4; ++i)
    {
        cout << sums[i] << endl;
    }
}