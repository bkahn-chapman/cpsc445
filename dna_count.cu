
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <cmath>
#include <array>
using namespace std;

__global__
void count(int *b, int *c, int N) {
    __syncthreads();
    int i = blockIdx.x;
    __syncthreads();
    if (i<N) {
        int t = b[i];
        __syncthreads();
        c[t]++; 
        __syncthreads();
    }
}

int main () {
    ifstream inFS;
    inFS.open("dna.txt");
    string dna;
    getline(inFS, dna);
    inFS.close();
    int N = dna.length();
    char ha[N];
    int hb[N];
    int hc[4];
    int *db;
    int *dc;
    hipMalloc((void **)&db, N*sizeof(int));
    hipMalloc((void **)&dc, 4*sizeof(int));
    for (int i = 0; i<N; ++i) {
        ha[i] = dna[i];
    }
    for(int i = 0; i<N; ++i) {
        if(ha[i] == 'A')
        {
            hb[i] = 0;
        }
        if(ha[i] == 'T')
        {
            hb[i] = 1;
        }
        if(ha[i] == 'G')
        {
            hb[i] = 2;
        }
        if(ha[i] == 'C')
        {
            hb[i] = 3;
        }
    }
    for(int i = 0; i < 4; ++i)
    {
        hc[i] = 0;
    }
    hipMemcpy(db, hb, N*sizeof(int), hipMemcpyHostToDevice);
    count<<<N, 1>>>(db, dc, N);
    hipMemcpy(hc, dc, 4*sizeof(int), hipMemcpyDeviceToHost);
    ofstream outFS;
    outFS.open("output.txt");
    outFS << "A " << hc[0] << endl;
    outFS << "T " << hc[1] << endl;
    outFS << "G " << hc[2] << endl;
    outFS << "C " << hc[3] << endl;
    outFS.close();
    hipFree(db);
    hipFree(dc);
    return 0;
}
