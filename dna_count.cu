
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <cmath>
#include <array>
using namespace std;

__global__
void count(char *a, int *b, int N) {
    int i = blockIdx.x;
    if (i<N) {
        if(a[i] == 'A')
        {
            b[0]++;
        }
        if(a[i] == 'C')
        {
            b[1]++;
        }
        if(a[i] == 'G')
        {
            b[2]++;
        }
        if(a[i] == 'T')
        {
            b[3]++;
        }
    }
}

int main () {
    ifstream inFS;
    inFS.open("dna.txt");
    string dna;
    getline(inFS, dna);
    inFS.close();
    int N = dna.length();
    char ha[N];
    int hb[4];
    char *da;
    int *db;
    hipMalloc((void **)&da, N*sizeof(char));
    hipMalloc((void **)&db, 4*sizeof(int));
    for (int i = 0; i<N; ++i) {
        ha[i] = dna[i];
    }
    for (int i = 0; i < 4; ++i) {
        hb[i] = 0;
    }
    hipMemcpy(da, ha, N*sizeof(char), hipMemcpyHostToDevice);
    int W = 4;
    count<<<1, W>>>(da, db, N);
    hipDeviceSynchronize();
    int sums[4];
    hipMemcpy(sums, db, 4*sizeof(int), hipMemcpyDeviceToHost);
    int sum = 0;
    for(int i = 0; i < W; ++i)
    {
        sums[i] += sums[i];
    }
    ofstream outFS;
    outFS.open("output.txt");
    char letters[] = {'A', 'C', 'G', 'T'};
    for(int i = 0; i<4; ++i)
    {
      outFS << letters[i] <<  " " << hb[i] << endl;
    }
    outFS.close();
    hipFree(da);
    hipFree(db);
    return 0;
}