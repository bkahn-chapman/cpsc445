#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <cmath>
#include <array>
using namespace std;

//test

__global__
void count(char *a, int *b, int N) {
    int i = blockIdx.x;
    __shared__ int share[4];
    int test[4];
    for(int c = 0; c < 4; ++c)
    {
        test[c] = 0;
    }
    if (i<N) {
        if(a[i] == 'A')
        {
            test[0]++;
        }
        if(a[i] == 'C')
        {
            test[1]++;
        }
        if(a[i] == 'G')
        {
            test[2]++;
        }
        if(a[i] == 'T')
        {
            test[3]++;
        }
    }
    __syncthreads();
    cout << test[0] << " " << test[1] << " " << test[2] << " " << test[3] << endl;
    for(int t = 0; t < 4; ++t)
    {
        b[t] = share[t];
    }
}

int main () {
    ifstream inFS;
    inFS.open("dna.txt");
    string dna;
    getline(inFS, dna);
    inFS.close();
    int N = dna.length();
    char ha[N];
    int hb[4];
    char *da;
    int *db;
    hipMalloc((void **)&da, N*sizeof(char));
    hipMalloc((void **)&db, N*sizeof(int));
    for (int i = 0; i<N; ++i) {
        ha[i] = dna[i];
    }
    hipMemcpy(da, ha, N*sizeof(char), hipMemcpyHostToDevice);
    count<<<N, 1>>>(da, db, N);
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);
    ofstream outFS;
    outFS.open("output.txt");
    for(int i = 0; i<4; ++i)
    {
      outFS << hb[i];
    }
    outFS.close();
    hipFree(da);
    hipFree(db);
    return 0;
}