#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <cmath>
#include <array>
using namespace std;

__global__
void count(char *a, int *b, int N) {
    int i = blockIdx.x;
    if (i<N) {
        if(a[i] == 'A')
        {
            b[0]++;
        }
        if(a[i] == 'C')
        {
            b[1]++;
        }
        if(a[i] == 'G')
        {
            b[2]++;
        }
        if(a[i] == 'T')
        {
            b[3]++;
        }
    }
}

int main () {
    ifstream inFS;
    inFS.open("dna.txt");
    string dna;
    getline(inFS, dna);
    inFS.close();
    int N = dna.length();
    char ha[N]
    int hb[4];
    char *da
    int *db;
    hipMalloc((void **)&da, N*sizeof(char));
    hipMalloc((void **)&db, N*sizeof(int));
    for (int i = 0; i<N; ++i) {
        ha[i] = dna[i];
    }
    hipMemcpy(da, ha, N*sizeof(char), hipMemcpyHostToDevice);
    invert<<<N, 1>>>(da, db, N);
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);
    ofstream outFS;
    outFS.open("output.txt");
    for(int i = 0; i<N; ++i)
    {
      outFS << hb[i];
    }
    outFS.close();
    hipFree(da);
    hipFree(db);
    return 0;
}