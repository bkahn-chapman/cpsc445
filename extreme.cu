#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <array>
#include <string>
using namespace std;

__global__
void extreme(int *a, int *b, int c, int r, int N)
{

}

int main () {
    ifstream inFS;
    inFS.open("input.csv");
    //system("head input.csv");
    string num;
    vector<int> nums;
    int rowcount = 0;
    int colcount = 1;
    string line;
    while(getline(inFS, line))
    {
        if(rowcount == 0)
        {
            for(char& d : line)
            {
                if(d == ',')
                {
                    colcount++;
                }
            }
        }
        rowcount++;
    }
    char c;
    ifstream if2;
    inFS.close();
    if2.open("input.csv");
    while(if2 >> noskipws >> c)
    {
        if(c == ',' || c == '\n')
        {
            nums.push_back(stoi(num));
            num = "";
        }
        else
        {
            num.push_back(c);
        }
    }
    int N = nums.size();
    int ha[N], hb[N];
    int *da, *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));
    for(int i = 0; i < N; ++i)
    {
        ha[i] = nums[i];
        hb[i] = 0;
    }
    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
    extreme<<<N, 1>>>(da, db, colcount, rowcount, N);
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyHostToDevice);
    for(int i = 0; i < hb.size(); ++i)
    {
        cout << hb[i] << endl;
    }
}