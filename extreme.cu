
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <array>
#include <string>
using namespace std;

int main () {
    ifstream inFS;
    inFS.open("input.csv");
    //system("head input.csv");
    string num;
    //vector<int> nums;
    int rowcount = 0;
    int colcount = 1;
    string line;
    while(getline(inFS, line))
    {
        //cout << line << endl;
        if(rowcount == 0)
        {
            for(char& d : line)
            {
                if(d == ',')
                {
                    colcount++;
                }
            }
        }
        rowcount++;
    }
    cout << colcount << endl;
    cout << rowcount << endl;
    int nums[colcount][rowcount];
    for(int i = 0; i < colcount; ++i)
    {
        for(int j = 0; j < rowcount; ++j)
        {
            nums[i][j] = 0;
        }
    }
    int charcount = 0;
    char c;
    inFS.close();
    inFS.open("input.csv");
    while(inFS >> noskipws >> c)
    {
        if(c != ',' && c != '\n')
        {
            num.push_back(c);
        }
        else
        {
            nums[charcount/colcount][charcount%colcount] = stoi(num);
            num = "";
        }
        charcount++;
    }
    cout << endl;
    /*
    for(int i = 0; i < colcount; ++i)
    {
        for(int j = 0; j < rowcount; ++j)
        {
            cout << nums[i][j] << ",";
        }
        cout << endl;
    }
    */
}