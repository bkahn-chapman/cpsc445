
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <array>
#include <string>
using namespace std;

__global__
void extreme(int *a, int *b, int N)
{
    int i = blockIdx.x;
    if (i<N) {
        b[i] = a[i]; 
        //no idea why a[i] isn't properly being accessed
        //plan was to have it check the eight surrounding and put a 1 in the associated slot in b
        //main would then determine which coordinate is at each value in b using the total number of rows and columns
        //cannot progress further though if this code in extreme isn't working as it did in sqrt and others
    }
}

int main () {
    ifstream inFS;
    inFS.open("input.csv");
    //system("head input.csv");
    string num;
    vector<int> nums;
    int rowcount = 0;
    int colcount = 1;
    string line;
    while(getline(inFS, line))
    {
        if(rowcount == 0)
        {
            for(char& d : line)
            {
                if(d == ',')
                {
                    colcount++;
                }
            }
        }
        rowcount++;
    }
    char c;
    ifstream if2;
    inFS.close();
    if2.open("input.csv");
    while(if2 >> noskipws >> c)
    {
        if(c == ',' || c == '\n')
        {
            nums.push_back(stoi(num));
            num = "";
        }
        else
        {
            num.push_back(c);
        }
    }
    int N = nums.size();
    cout << N << endl;
    int ha[N], hb[N];
    int *da, *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));
    for(int i = 0; i < N; ++i)
    {
        ha[i] = nums[i];
    }
    /*
    for(int i = 0; i < N; ++i)
    {
        cout << ha[i] << endl;
    }
    */
    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
    extreme<<<N, 1>>>(da, db, N);
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyHostToDevice);
    for(int i = 0; i < N; ++i)
    {
        cout << hb[i] << endl;
    }
}