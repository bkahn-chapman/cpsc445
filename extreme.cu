
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <array>
#include <string>
using namespace std;

int main () {
    ifstream inFS;
    inFS.open("input.csv");
    //system("head input.csv");
    string num;
    int rowcount = 0;
    int colcount = 1;
    string line;
    while(getline(inFS, line))
    {
        cout << line << endl;
        if(rowcount == 0)
        {
            for(char& d : line)
            {
                if(d == ',')
                {
                    colcount++;
                }
            }
        }
        rowcount++;
    }
    double nums[colcount][rowcount];
    int charcount = 0;
    char c;
    while(inFS >> noskipws >> c)
    {
        if(c != ',' && c != '\n')
        {
            num.push_back(c);
        }
        else
        {
            cout << num << endl;
            nums[charcount/colcount][charcount%colcount] = atof(num.c_str());
            num = "";
        }
        charcount++;
    }
    cout << endl;
    /*
    for(int i = 0; i < colcount; ++i)
    {
        for(int j = 0; j < rowcount; ++j)
        {
            cout << nums[i][j] << ",";
        }
        cout << endl;
    }
    */
}