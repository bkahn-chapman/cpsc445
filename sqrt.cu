#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <array>
#include <string>
using namespace std;

__global__
void squareroot(double *a, double *b, int N) {
    int i = blockIdx.x;
    if (i<N) {
        b[i] = sqrt(a[i]);
    }
}

int main () {
    ifstream inFS;
    inFS.open("input.csv");
    //system("head input.csv");
    vector<double> nums;
    string line;
    while(getline(inFS, line))
    {
        string num = "";
        for(char &c : line)
        {
            if(c == '\n')
            {
                nums.push_back(stod(num));
            }  
            else
            {
                num.push_back(c);
            }
        }
        cout << atof(num) << endl;
    }
    int N = nums.size();
    cout << N << endl;
    double ha[N], hb[N];
    double *da, *db;
    hipMalloc((void **)&da, N*sizeof(double));
    hipMalloc((void **)&db, N*sizeof(double));
    for (int i = 0; i<N; ++i) {
        ha[i] = nums[i];
    }
    hipMemcpy(da, ha, N*sizeof(double), hipMemcpyHostToDevice);
    squareroot<<<N, 1>>>(da, db, N);
    hipMemcpy(hb, db, N*sizeof(double), hipMemcpyDeviceToHost);
    ofstream outFS;
    outFS.open("output.csv");
    for(int i = 0; i<N; ++i)
    {
      cout << hb[i] << endl;
      outFS << hb[i] << endl;
    }
    outFS.close();
    hipFree(da);
    hipFree(db);
    return 0;
}