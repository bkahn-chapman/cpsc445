
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <cmath>
#include <array>
#include <string>
#include <algorithm>
using namespace std;

int main () {
    ifstream inFS;
    inFS.open("dna.txt");
    string dna;
    getline(inFS, dna);
    cout << dna.length() << endl;
    int N = dna.length() / 3;
    string ha[N];
    string triplet;
    for(int i = 0; i < N; ++i)
    {
        triplet.push_back(dna[i]);
        triplet.push_back(dna[i+1]);
        triplet.push_back(dna[i+2]);
        ha[i] = triplet;
        triplet = "";
    }
    for(int i = 0; i < N; ++i)
    {
        cout << triplet[i] << endl;
    }
}