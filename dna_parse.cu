
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <cmath>
#include <array>
#include <string>
#include <algorithm>
using namespace std;

__global__
void parse(int *b, int *c, int N) {
    int i = blockIdx.x;
    if (i<N) {
        int t = b[i];
        c[t]++; //race condition? sometimes?
    }
}

int main () {
    ifstream inFS;
    inFS.open("dna.txt");
    string dna;
    getline(inFS, dna);
    int N = dna.length() / 3;
    string ha[N];
    int hb[N];
    int hc[64];
    int *db;
    int *dc;
    hipMalloc((void **)&db, N*sizeof(int));
    hipMalloc((void **)&dc, 64*sizeof(int));
    string triplet;
    for(int i = 0; i < N; ++i)
    {
        triplet.push_back(dna[3*i]);
        triplet.push_back(dna[3*i+1]);
        triplet.push_back(dna[3*i+2]);
        ha[i] = triplet;
        triplet = "";
    }
    char letts[] = {'A', 'C', 'G', 'T'};
    string trips = "";
    for(int t = 0; t < N; ++t)
    {
        int count = 0;
        for(int i = 0; i < 4; ++i)
        {
            for(int j = 0; j < 4; ++j)
            {
                for(int k = 0; k < 4; ++k)
                {
                    trips = "";
                    trips.push_back(letts[i]);
                    trips.push_back(letts[j]);
                    trips.push_back(letts[k]);
                    if(trips == ha[t])
                    {
                        hb[t] = count;
                    }
                    count++;
                }
            }
        }
    }
    for(int i = 0; i < 64; ++i)
    {
        hc[i] = 0;
    }
    hipMemcpy(db, hb, N*sizeof(int), hipMemcpyHostToDevice);
    parse<<<N, 1>>>(db, dc, N);
    hipMemcpy(hc, dc, 64*sizeof(int), hipMemcpyDeviceToHost);
    ofstream outFS;
    outFS.open("output.txt");
    int count = 0;
    for(int i = 0; i < 4; ++i)
    {
        for(int j = 0; j < 4; ++j)
        {
            for(int k = 0; k < 4; ++k)
            {
                trips = "";
                trips.push_back(letts[i]);
                trips.push_back(letts[j]);
                trips.push_back(letts[k]);
                if(hc[count] > 0)
                {
                    outFS << trips << " " << hc[count] << endl;
                }
                count++;
            }
        }
    }
}