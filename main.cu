#include "hip/hip_runtime.h"
#include <string>
#include <fstream>
#include <iostream>
#include <vector>
using namespace std;

//the function to find the overlaps
//COULD NOT GET THIS TO WORK WITH THE SETUP I CREATED
//plan was to have one shape at a time check for overlaps
//it would check if any point in the shape is contained in the min/max square of the other shapes
//if there was an overlap, add the shapes to the overlap array (ex. 1 2 0 0 1 4 0 0 2 3 0 0 0 0 0 0 3 5 4 5)
//then return that array for main to handle the logic of the array and output results
__global__
void getOverlaps(double *m, double *s, double *n, double *o, int M, int N, int S, int O)
{
    int i = blockIdx.x;
    if(i < N)
    {
        o[i] = 1;
    }
}

int main () {
    ifstream inFS;
    inFS.open("input.txt");
    vector<double> max_min; //each row's max and min X and Y values
    vector<double> sizes; //the number of points in each shape
    vector<double> nums; //every single point of every single shape
    vector<double> overlaps; //contains a variable for every potential overlap
    string line; //each line of the file
    while(getline(inFS, line))
    {
        string numX; //the point's X value
        string numY; //the point's Y value
        double max_x = 0; //the shape's max X
        double min_x = 0; //the shape's min X
        double max_y = 0; //the shape's max Y
        double min_y = 0; //the shape's min Y
        int checkFirstX = 0; //checks if the point is the first in the shape
        int checkFirstY = 0; //checks if the point is the first in the shape
        int XorY = 0; //checks if it is the X variable or the Y variable
        int pairs = 0; //tracks how many points are in each shape
        for(int i = 0; i < line.length()-1; ++i) //checks each point in the shape
        {
            if(line[i] == '(') //if it's the start of the point
            {
                for(int t = i+1; t < line.length(); ++t) //each char in the point
                {
                    if(line[t] == ',') //if the X variable of the point is finished
                    {
                        if(checkFirstX == 0) //if the first point in the shape
                        {
                            max_x = atof(numX.c_str()); //sets max X
                            min_x = atof(numX.c_str()); //sets min X
                            nums.push_back(max_x); //adds the X variable to the list
                            checkFirstX = 1; //ends the check
                        }
                        else
                        {
                            double testX = atof(numX.c_str()); //the X variable that isn't from the first point
                            if(testX > max_x) //if higher than the current max
                            {
                                max_x = testX; //new max
                            }
                            else if(testX < min_x) //if lower than the current min
                            {
                                min_x = testX; //new min
                            }
                            nums.push_back(testX); //adds the X variable to the list
                        }
                        numX = ""; //resets the number string
                        XorY = 1; //next number will be a Y
                    }
                    else if(line[t] == ')') //if the Y variable of the point is finished
                    {
                        if(checkFirstY == 0) //if the first point in the shape
                        {
                            max_y = atof(numY.c_str()); //sets max Y
                            min_y = atof(numY.c_str()); //sets min Y
                            nums.push_back(max_y); //adds the Y variable to the list
                            checkFirstY = 1; //ends the check
                        }
                        else
                        {
                            double testY = atof(numY.c_str());; //the Y variable that isn't from the first point
                            if(testY > max_y) //if higher than the current max
                            {
                                max_y = testY; //new max
                            }
                            else if(testY < min_y) //if lower than the current min
                            {
                                min_y = testY; //new min
                            }
                            nums.push_back(testY); //adds the Y variable to the list
                        }
                        numY = ""; //resets the number string
                        XorY = 0; //next number will be an X
                        pairs++; //counts that another point has been finished
                        break; //ends the loop for the point
                    }
                    else //if there is still more to the variable
                    {
                        if(XorY == 0) //if X variable
                        {
                            numX.push_back(line[t]); //add the next char to the X variable
                        }
                        else //if Y variable
                        {
                            numY.push_back(line[t]); //add the next char to the Y variable
                        }
                    }
                }
            }
        }
        sizes.push_back(pairs); //adds the number of points in the shape to the list
        max_min.push_back(max_x); //adds the max X of the shape
        max_min.push_back(min_x); //adds the min X of the shape
        max_min.push_back(max_y); //adds the max Y of the shape
        max_min.push_back(min_y); //adds the min Y of the shape
    }
    inFS.close();
    int M = max_min.size(); //the total number of max/min
    int N = nums.size(); //the total number of points * 2
    int S = sizes.size(); //the total number of shapes
    //cout << M << endl;
    //cout << N << endl;
    //cout << S << endl;
    int p = 0; //number of possible overlap pairings
    for(int i = 1; i < S; ++i) //based on the number of shapes
    {
        p = p + i; //(shapes - 1) + (shapes - 2)... + 1
    }
    int O = p*2; //number of possible overlaps * 2 (one slot for each shape in the overlap)
    double hm[M], hn[N], hs[S], hO[O]; //arrays for each vector (m = min-max, n = nums, s = sizes, o =overlaps)
    double *dm, *dn, *ds, *dO; //pointers for each array 
    hipMalloc((void **)&dm, M*sizeof(double)); //size of min-max array
    hipMalloc((void **)&dn, N*sizeof(double)); //size of nums array
    hipMalloc((void **)&ds, S*sizeof(double)); //size of sizes array
    hipMalloc((void **)&dO, O*sizeof(double)); //size of overlaps array
    //fill the nums array
    for (int i = 0; i<N; ++i) {
        hn[i] = nums[i];
    }
    //fill the min-max array
    for (int i = 0; i<M; ++i) {
        hm[i] = max_min[i];
    }
    //fill the sizes array
    for (int i = 0; i<S; ++i) {
        hs[i] = sizes[i];
    }
    //fill the overlaps array
    for (int i = 0; i<O; ++i) {
        hO[i] = 0;
    }
    hipMemcpy(dm, hm, M*sizeof(double), hipMemcpyHostToDevice); //sends min-max array to device
    hipMemcpy(dn, hn, N*sizeof(double), hipMemcpyHostToDevice); //sends nums array to device
    hipMemcpy(ds, hs, S*sizeof(double), hipMemcpyHostToDevice); //sends sizes array to device
    hipMemcpy(dO, hO, O*sizeof(double), hipMemcpyHostToDevice); //sends overlaps array to device
    getOverlaps<<<N, 1>>>(dm, ds, dn, dO, M, S, N, O); //the global function to find the overlaps
    hipMemcpy(hO, dO, O*sizeof(double), hipMemcpyDeviceToHost); //receives the overlaps array back to the host
    ofstream outFS;
    outFS.open("output.txt")
    for(int i = 0; i < O; ++i) //iterates through the entire received overlaps array
    {
        if(hO[i] != 0) //if the overlap checker isn't empty
        {
            if(i % 2 == 0) //if the first of a pair of shapes (0 to 1, 4 to 5, etc.)
            {
                outFS << "Shape: " << ho[i] << " overlaps with ";
            }
            else //if the second of a pair of shapes (1 with 0, 5 with 4, etc.)
            {
                outFS << "shape: " << ho[i] << "." << endl;
            }
        }

    }
}
