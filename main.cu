#include "hip/hip_runtime.h"
#include <string>
#include <fstream>
#include <iostream>
#include <vector>
using namespace std;

extern vector<int> corners;

__global__
void find_corners(int N)
{
    corners.push_back(N);
}

int main () {
    ifstream inFS;
    inFS.open("input.txt");
    vector<string> polygons;
    string line;
    while(getline(inFS, line))
    {
        polygons.push_back(line);
    }
    int N = polygons.size();
    hipMemcpy(N, hipMemcpyHostToDevice);
    find_corners<<<N, 1>>>(N);
    hipMemcpy(N, hipMemcpyDeviceToHost);
}