#include "hip/hip_runtime.h"
#include <string>
#include <fstream>
#include <iostream>
#include <vector>
using namespace std;

__global__
void find_corners(string *a, int *b, int N)
{
    
}

int main () {
    ifstream inFS;
    inFS.open("input.txt");
    vector<double> max_min;
    string line;
    while(getline(inFS, line))
    {
        string numX;
        string numY;
        double max_x = 0;
        double min_x = 0;
        double max_y = 0;
        double min_y = 0;
        int checkFirstX = 0;
        int checkFirstY = 0;
        int XorY = 0;
        for(int i = 0; i < line.length()-1; ++i)
        {
            if(line[i] == '(')
            {
                for(int t = i; t < line.length()-1; ++t)
                {
                    if(t[i] == ',')
                    {
                        if(checkFirstX == 0)
                        {
                            max_x = stod(numX);
                            min_x = stod(numX);
                            checkFirstX = 1;
                        }
                        else
                        {
                            if(stod(numX) > max_x)
                            {
                                max_x = stod(numX);
                            }
                            else if(stod(numX) < min_x))
                            {
                                min_x = stod(numX);
                            }
                        }
                        numX = "";
                        XorY = 1;
                    }
                    else if(t[i] == ')')
                    {
                        if(checkFirstY == 0)
                        {
                            max_y = stod(numY);
                            min_y = stod(numY);
                            checkFirstY = 1;
                        }
                        else
                        {
                            if(stod(numY) > max_y)
                            {
                                max_y = stod(numY);
                            }
                            else if(stod(numY) < min_y))
                            {
                                min_y = stod(numY);
                            }
                        }
                        numY = "";
                        XorY = 0;
                    }
                    else
                    {
                        if(XorY == 0)
                        {
                            numX.append(t[i]);
                        }
                        else
                        {
                            numY.append(t[i]);
                        }
                    }
                }
            }
        }
    }
}