#include "hip/hip_runtime.h"
#include <string>
#include <fstream>
#include <iostream>
#include <vector>
using namespace std;

extern __shared__ vector<int> corners;

__global__
void find_corners(int N)
{
    corners.push_back(N);
}

int main () {
    ifstream inFS;
    inFS.open("input.txt");
    vector<string> polygons;
    string line;
    while(getline(inFS, line))
    {
        polygons.push_back(line);
    }
    int N = polygons.size();
    find_corners<<<N, 1>>>(N);
}