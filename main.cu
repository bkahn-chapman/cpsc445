#include "hip/hip_runtime.h"
#include <string>
#include <fstream>
#include <iostream>
#include <vector>
using namespace std;

__global__
void find_corners(string *a, int *b, int N)
{
    
}

int main () {
    ifstream inFS;
    inFS.open("input.txt");
    vector<string> polygons;
    string line;
    while(getline(inFS, line))
    {
        polygons.push_back(line);
    }
    int N = polygons.size();
    vector<string> ha;
    double hb[N*4];
    vector<string> *da;
    double *db;
    hipMalloc((void **)&da, N*sizeof(string));
    hipMalloc((void **)&db, N*sizeof(int));
    for(int i = 0; i<N; ++i)
    {
        ha[i] = polygons[i];
    }
    cout << ha[i] << endl;
    hipMemcpy(da, ha, N*sizeof(string), hipMemcpyHostToDevice);
    find_corners<<<N, 1>>>(da, db, N);
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);
}