#include "hip/hip_runtime.h"
#include <string>
#include <fstream>
#include <iostream>
#include <vector>
using namespace std;

__global__
void getOverlaps(double *m, double *s, double *n, double *o, int S)
{

}

int main () {
    ifstream inFS;
    inFS.open("input.txt");
    vector<double> max_min;
    vector<double> sizes;
    vector<double> nums;
    vector<double> overlaps;
    string line;
    while(getline(inFS, line))
    {
        string numX;
        string numY;
        double max_x = 0;
        double min_x = 0;
        double max_y = 0;
        double min_y = 0;
        int checkFirstX = 0;
        int checkFirstY = 0;
        int XorY = 0;
        int pairs = 0;
        for(int i = 0; i < line.length()-1; ++i)
        {
            if(line[i] == '(')
            {
                for(int t = i+1; t < line.length(); ++t)
                {
                    if(line[t] == ',')
                    {
                        if(checkFirstX == 0)
                        {
                            max_x = atof(numX.c_str());
                            min_x = atof(numX.c_str());
                            nums.push_back(max_x);
                            checkFirstX = 1;
                        }
                        else
                        {
                            double testX = atof(numX.c_str());
                            if(testX > max_x)
                            {
                                max_x = testX;
                            }
                            else if(testX < min_x)
                            {
                                min_x = testX;
                            }
                            nums.push_back(testX);
                        }
                        numX = "";
                        XorY = 1;
                    }
                    else if(line[t] == ')')
                    {
                        if(checkFirstY == 0)
                        {
                            max_y = atof(numY.c_str());
                            min_y = atof(numY.c_str());
                            nums.push_back(max_y);
                            checkFirstY = 1;
                        }
                        else
                        {
                            double testY = atof(numY.c_str());;
                            if(testY > max_y)
                            {
                                max_y = testY;
                            }
                            else if(testY < min_y)
                            {
                                min_y = testY;
                            }
                            nums.push_back(testY);
                        }
                        numY = "";
                        XorY = 0;
                        pairs++;
                        break;
                    }
                    else
                    {
                        if(XorY == 0)
                        {
                            numX.push_back(line[t]);
                        }
                        else
                        {
                            numY.push_back(line[t]);
                        }
                    }
                }
            }
        }
        sizes.push_back(pairs);
        max_min.push_back(max_x);
        max_min.push_back(min_x);
        max_min.push_back(max_y);
        max_min.push_back(min_y);
    }
    inFS.close();
    int M = max_min.size();
    int N = nums.size();
    int S = sizes.size();
    int p = 0;
    for(int i = 1; i < S; ++i)
    {
        p = p + i;
    }
    cout << p << endl;
    int O = overlaps.size();
    double hm[M], hn[N], hs[S], hO[O];
    double *dm, *dn, *ds, *dO;
    hipMalloc((void **)&dm, N*sizeof(double));
    hipMalloc((void **)&dn, N*sizeof(double));
    hipMalloc((void **)&ds, N*sizeof(double));
    hipMalloc((void **)&dO, N*sizeof(double));
    for (int i = 0; i<N; ++i) {
        hn[i] = nums[i];
    }
    for (int i = 0; i<M; ++i) {
        hm[i] = max_min[i];
    }
    for (int i = 0; i<S; ++i) {
        hs[i] = sizes[i];
    }
    for (int i = 0; i<O; ++i) {
        hO[i] = overlaps[i];
    }
    hipMemcpy(dm, hm, M*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dn, hn, N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(ds, hs, S*sizeof(double), hipMemcpyHostToDevice);
    getOverlaps<<<S, 1>>>(dm, ds, dn, S);
}
