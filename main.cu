
#include <hip/hip_runtime.h>
#include <string>
#include <fstream>
#include <iostream>
#include <vector>
using namespace std;

__global__
void find_corners(string *a, int *b, int N)
{
    
}

int main () {
    ifstream inFS;
    inFS.open("input.txt");
    vector<double> max_min;
    vector<double> sizes;
    string line;
    while(getline(inFS, line))
    {
        string numX;
        string numY;
        double max_x = 0;
        double min_x = 0;
        double max_y = 0;
        double min_y = 0;
        int checkFirstX = 0;
        int checkFirstY = 0;
        int XorY = 0;
        int pairs = 0;
        for(int i = 0; i < line.length()-1; ++i)
        {
            if(line[i] == '(')
            {
                for(int t = i+1; t < line.length(); ++t)
                {
                    if(line[t] == ',')
                    {
                        if(checkFirstX == 0)
                        {
                            max_x = atof(numX.c_str());
                            min_x = atof(numX.c_str());
                            checkFirstX = 1;
                        }
                        else
                        {
                            double testX = atof(numX.c_str());
                            if(testX > max_x)
                            {
                                max_x = testX;
                            }
                            else if(testX < min_x)
                            {
                                min_x = testX;
                            }
                        }
                        numX = "";
                        XorY = 1;
                    }
                    else if(line[t] == ')')
                    {
                        if(checkFirstY == 0)
                        {
                            max_y = atof(numY.c_str());;
                            min_y = atof(numY.c_str());;
                            checkFirstY = 1;
                        }
                        else
                        {
                            double testY = atof(numY.c_str());;
                            if(testY > max_y)
                            {
                                max_y = testY;
                            }
                            else if(testY < min_y)
                            {
                                min_y = testY;
                            }
                        }
                        numY = "";
                        XorY = 0;
                        pairs++;
                        break;
                    }
                    else
                    {
                        if(XorY == 0)
                        {
                            numX.push_back(line[t]);
                        }
                        else
                        {
                            numY.push_back(line[t]);
                        }
                    }
                }
            }
        }
        sizes.push_back(pairs);
        max_min.push_back(max_x);
        max_min.push_back(min_x);
        max_min.push_back(max_y);
        max_min.push_back(min_y);
    }
    for(int i = 0; i < sizes.size(); ++i)
    {
        cout << sizes[i] << endl;
    }
}