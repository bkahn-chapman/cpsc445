#include "hip/hip_runtime.h"
#include <string>
#include <fstream>
#include <iostream>
#include <vector>
using namespace std;

__global__
void find_corners(string *a, int N)
{
    corners.push_back(N);
}

int main () {
    ifstream inFS;
    inFS.open("input.txt");
    vector<string> polygons;
    string line;
    while(getline(inFS, line))
    {
        polygons.push_back(line);
    }
    int N = polygons.size();
    vector<string> ha;
    vecor<string> *da;
    hipMalloc((void **)&da, N*sizeof(string));
    for(int i = 0; i<N; ++i)
    {
        ha[i] = polygons[i];
    }
    hipMemcpy(ha, N*sizeof(string), hipMemcpyHostToDevice);
    find_corners<<<N, 1>>>(ha, N);
    hipMemcpy(hb, N, hipMemcpyDeviceToHost);
}